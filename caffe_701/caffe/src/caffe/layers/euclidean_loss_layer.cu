#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top) {
  
  Forward_cpu(bottom, top);
  return;

  printf("Forward_gpu 0\n");

  int N=bottom[0]->num(),
      C=bottom[0]->channels(),
      H=bottom[0]->height(),
      W=bottom[0]->width();
  int gtN=(int)bottom[1]->cpu_data()[0];
  
  printf("Forward_gpu 1\n");

  int inN=N-gtN;
  int CxHxW=C*H*W;
  caffe_gpu_set(inN,Dtype(0),sumexp1overL2dist_.mutable_gpu_data());

  printf("Forward_gpu 2\n");

  caffe_gpu_set(gtN*inN, Dtype(1), all1s_.mutable_gpu_data());

  printf("Forward_gpu 3\n");

  // compute diff and L2dist
  for(int i=0;i<inN;i++){
    for(int j=0;j<gtN;j++){
      caffe_gpu_sub(CxHxW,
        bottom[0]->gpu_data()+(gtN+i)*CxHxW, // f(i)
        bottom[0]->gpu_data()+j*CxHxW, // f(j)
        diff_.mutable_gpu_data()+(i*gtN+j)*CxHxW); 
      caffe_gpu_dot(CxHxW,
        diff_.gpu_data()+(i*gtN+j)*CxHxW,
        diff_.gpu_data()+(i*gtN+j)*CxHxW,
        L2dist_.mutable_gpu_data()+i*gtN+j);
    }
  }

  printf("Forward_gpu 4\n");

  // compute exp(1/L2dist)
  caffe_gpu_div(gtN*inN,all1s_.gpu_data(),L2dist_.gpu_data(),exp1overL2dist_.mutable_gpu_data());

  printf("Forward_gpu 5\n");


  caffe_gpu_exp(gtN*inN,exp1overL2dist_.gpu_data(),exp1overL2dist_.mutable_gpu_data());

  printf("Forward_gpu 6\n");

  // compute sum(exp(1/L2dist))
  for(int i=0;i<inN;i++)
    for(int j=0;j<gtN;j++)
      sumexp1overL2dist_.mutable_gpu_data()[i]+=exp1overL2dist_.gpu_data()[i*gtN+j];
  Dtype LOSS=Dtype(0);

  printf("Forward_gpu 7\n");

  // for each input frame f(i)
  for(int i=0;i<inN;i++){
    // compute LOSSi
    Dtype LOSSi=Dtype(0);
    for(int j=0;j<gtN;j++)
      LOSSi+=L2dist_.gpu_data()[i*gtN+j]*exp1overL2dist_.gpu_data()[i*gtN+j]/sumexp1overL2dist_.gpu_data()[i];
    // add to total LOSS
    LOSS+=LOSSi;
  }
  top[0]->mutable_cpu_data()[0]=LOSS;

  printf("Forward_gpu 8\n");
}

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  
  Backward_cpu(top, propagate_down, bottom);
  return;


  const Dtype loss_weight=top[0]->cpu_diff()[0];
  int N=bottom[0]->num(),
      C=bottom[0]->channels(),
      H=bottom[0]->height(),
      W=bottom[0]->width();
  int gtN=(int)bottom[1]->cpu_data()[0];
  int inN=N-gtN;
  int CxHxW=C*H*W;
  caffe_gpu_set(N*CxHxW,Dtype(0),bottom[0]->mutable_gpu_diff());
  // ===== for each GT frame f(j) ===== 
  for(int j=0;j<gtN;j++){
    Dtype* bottom_diff_j=bottom[0]->mutable_gpu_diff()+j*CxHxW;
    for(int i=0;i<inN;i++){
      // compute 1st term
      caffe_gpu_axpby(CxHxW,
        Dtype(-2)*exp1overL2dist_.gpu_data()[i*gtN+j]/sumexp1overL2dist_.gpu_data()[i],
        diff_.gpu_data()+(i*gtN+j)*CxHxW,
        Dtype(1),bottom_diff_j);
      // compute 2nd term
      caffe_gpu_axpby(CxHxW,
        exp1overL2dist_.gpu_data()[i*gtN+j]*Dtype(2)/L2dist_.gpu_data()[i*gtN+j]/sumexp1overL2dist_.gpu_data()[i],
        diff_.gpu_data()+(i*gtN+j)*CxHxW,
        Dtype(1),bottom_diff_j);
      // compute 3rd term
      caffe_gpu_axpby(CxHxW,
        exp1overL2dist_.gpu_data()[i*gtN+j]*exp1overL2dist_.gpu_data()[i*gtN+j]*Dtype(-2)
          /L2dist_.gpu_data()[i*gtN+j]/sumexp1overL2dist_.gpu_data()[i]/sumexp1overL2dist_.gpu_data()[i],
        diff_.gpu_data()+(i*gtN+j)*CxHxW,
        Dtype(1),bottom_diff_j);
    }
  }
  // ===== for each input frame f(i) ===== 
  for(int i=0;i<inN;i++){
    Dtype* bottom_diff_i=bottom[0]->mutable_gpu_diff()+(gtN+i)*CxHxW;
    // compute 1st term
    for(int j=0;j<gtN;j++){
      caffe_gpu_axpby(CxHxW,
        Dtype(2)*exp1overL2dist_.gpu_data()[i*gtN+j]/sumexp1overL2dist_.gpu_data()[i],
        diff_.gpu_data()+(i*gtN+j)*CxHxW,
        Dtype(1),bottom_diff_i);
    }
    // compute 2nd term
    for(int j=0;j<gtN;j++){
      caffe_gpu_axpby(CxHxW,
        exp1overL2dist_.gpu_data()[i*gtN+j]*Dtype(-2)/L2dist_.gpu_data()[i*gtN+j]/sumexp1overL2dist_.gpu_data()[i],
        diff_.gpu_data()+(i*gtN+j)*CxHxW,
        Dtype(1),bottom_diff_i);
    }
    // compute 3rd term
    Dtype scale3rdterm=Dtype(0);
    for(int j=0;j<gtN;j++)
      scale3rdterm+=L2dist_.gpu_data()[i*gtN+j]*exp1overL2dist_.gpu_data()[i*gtN+j];
    scale3rdterm*=Dtype(-1)/sumexp1overL2dist_.gpu_data()[i]/sumexp1overL2dist_.gpu_data()[i];
    for(int j=0;j<gtN;j++){
      caffe_gpu_axpby(CxHxW,
        Dtype(-2)*exp1overL2dist_.gpu_data()[i*gtN+j]/L2dist_.gpu_data()[i*gtN+j]/L2dist_.gpu_data()[i*gtN+j]*scale3rdterm,
        diff_.gpu_data()+(i*gtN+j)*CxHxW,
        Dtype(1),bottom_diff_i);
    }
  }
  caffe_gpu_scal(N*CxHxW,loss_weight,bottom[0]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(EuclideanLossLayer);

}  // namespace caffe